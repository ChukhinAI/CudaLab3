
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

#define CHECK(value) {                                          \
    hipError_t _m_cudaStat = value;                                        \
    if (_m_cudaStat != hipSuccess) {                                       \
        cout<< "Error:" << hipGetErrorString(_m_cudaStat) \
            << " at line " << __LINE__ << " in file " << __FILE__ << "\n"; \
        exit(1);                                                            \
    } }

//calculate<<<(rowsY + 255) / 256, 256>>>(dev_original_matrix, dev_result, elementsQuantity, rowsY, colsX, border_number); <<<кол-во блоков в grid, кол-во потоков>>>
// + 255 для того, чтобы точно уместить все данные
// размер массива ограничен максимальным размером пространства потоков = 256
  __global__ void calculate(int *dev_original_matrix, int *dev_result, int elementsQuantity, int rowsY, int colsX, int border_number)
  {
    int current_row_number = threadIdx.x + blockIdx.x * blockDim.x; // номер строки в изображении
	// Grid -> блок -> поток(поток в блоке, блок в сетке), один поток запускает функцию calculate один раз
	// blockIdx - номер блока в 1D-grid, blockDim - кол-во блоков в одном потоке

	int r;
	int count_points = 0;
	for (int i = 1; i < colsX - 1; i++) // крайние не считаются
    {
        //r = matrix[i * colsX + j - 1] - matrix[i * colsX + j + 1]; // мб надо делить на 2, но в Гонсалесе вот так(см градиент Собела/Собеля);
		
		r = dev_original_matrix[current_row_number * colsX + i - 1] - dev_original_matrix[current_row_number * colsX + i + 1];
		if (r > border_number)
		{
			count_points++;
		}
    }
	dev_result[current_row_number] = count_points;
	
	
	//cout << "dev_result in GPU claculate :" << "\n";
	/*
	for (int i = 0; i < current_row_number; i++)
	{
		cout << dev_result[i] << " ";
	}
	cout << '\n';
	*/
}

//СОХРАНЯТЬ РЕЗУЛЬТАТ В ЛОК ПЕРЕМ ПОТОМ В РЕЗ МАТРИЦУ
__global__ void goodCalculation(int *dev_original_matrix, int *dev_result, int elementsQuantity,
									int rowsY, int colsX, int border_number)
{
	
	int rowsCountBeforeOrangeLine = blockIdx.x * blockDim.x;
    //int bigRowNumber = blockIdx.x * blockDim.x + threadIdx.x;

    int cacheWidth = 32;	 // original
    int rectangleHeight = 8; // original

    //int rectangleInRowQuantity = colsX / cacheWidth; // original
	int rectangleInRowQuantity = (colsX - 2) / (cacheWidth - 2);

    __shared__ int cache[256][33]; 

	int r;
	int count_points = 0;

	int rowInCache = threadIdx.x / cacheWidth;  // номер строки в верхнем ЗП (первый элемент)
	int currentRowInCache = rowInCache;
	int columnInCache = threadIdx.x % cacheWidth;
	int pixelCountUpperRowInTopGreenRect = (rowsCountBeforeOrangeLine + rowInCache) * colsX;	
	int indexTopPixelInCurrentFPInsideImage = pixelCountUpperRowInTopGreenRect + columnInCache;
	int verticalStep = rectangleHeight * colsX;	
    for (int stringIteration = 0; stringIteration < rectangleInRowQuantity; stringIteration++)
    {		
		int currentPixelInImage = indexTopPixelInCurrentFPInsideImage;
      for (int levelInCache = 0; levelInCache < cacheWidth; levelInCache++)
      {		
	    cache[currentRowInCache][columnInCache] = dev_original_matrix[currentPixelInImage]; 										 
        currentRowInCache += rectangleHeight; 
		currentPixelInImage += verticalStep; // verticalStep по ЗП вниз
       
      }
	  indexTopPixelInCurrentFPInsideImage += 30; // переход к след ФП
	  currentRowInCache = rowInCache;
      __syncthreads();
	  
	  r = 0;
	  
 // тут начинаются ошибки с неправильным обращенем к памяти - fixed
	  for (int i = 1; i < cacheWidth - 1; i++)
      {
		r = cache[threadIdx.x][i - 1] - cache[threadIdx.x][i + 1];
		if (r > border_number) // ошибка
			count_points = count_points + 1;		
      }

      __syncthreads();
    }

	dev_result[rowsCountBeforeOrangeLine + threadIdx.x] = count_points; // ошибка с неправильным обращенем к памяти - fixed
}

void printMatrix(int* matrix, int colsX, int rowsY)
{
  for (int i = 0; i < rowsY; i++)
  {
    for (int j = 0; j < colsX; j++)
    {
        cout << matrix[i * colsX + j] << "\t";
    }
    cout << "\n";
  }
}

bool checkResult(int* host_result, int* result, int colsX, int rowsY)
{

  for (int i = 0; i < 20; i++)
  {
	cout << "host_result[ " << i << " ] = " << host_result[i] << '\n';
	
  }
  
    for (int i = 0; i < 20; i++)
  {

	cout << "result[ " << i << " ] = " << result[i] << '\n';
  }
  
  for (int i = 0; i < rowsY; i++)
  {
    if (host_result[i] != result[i])
    {
	//cout << "host_result[ " << i << " ] = " << host_result[i] << '\n';
	//cout << "result[ " << i << " ] = " << result[i] << '\n';
      return false;
    }
  }

  return true;
}

int main(void)
{
    hipEvent_t startCUDA, stopCUDA, startOptimalCUDA, stopOptimalCUDA;
    clock_t startCPU;
    float elapsedUsualTimeCUDA, elapsedTimeCPU, elapsedOptimalTime;

    // 13. Создайте детектор вертикальных границ на изображении (в градациях серого). 
	// Функция должна для каждой строки считать количество точек, в которых производная цвета по горизонтали больше заданного значения.
	// Все изображения хранятся в памяти по строкам.

    int colsX = 1502; 		//  пикселей 30 * 50 + 2 = 1502
    int rowsY = 17920; 		//  пикселей 256 * 70 = 17920
    int elementsQuantity = colsX * rowsY;
    cout << "Size in Mbs = " << elementsQuantity * sizeof(int) / 1048576.0 << "\n";
    int *matrix = new int[elementsQuantity];

    for (int i = 0; i < rowsY; i++)
    {
      for (int j = 0; j < colsX; j++)
      {
          matrix[i * colsX + j] = rand() % 255; // filling matrix
		  //matrix[i * colsX + j] = (i * colsX + j) * 10 * i;
      }
    }

    //printMatrix(matrix, colsX, rowsY);

    int border_number = 10; // -410
	cout << "border_number = " << border_number << '\n';

    startCPU = clock();
    int *result = new int[rowsY];
    //int *count_points = new int[rowsY];
	int r;
	int count_points;
    for (int i = 0; i < rowsY; i++) // alg CPU func 
    {
      //int r = 0;
	  //int count_points = 0;
	  count_points = 0;

      for (int j = 1; j < colsX - 1; j++)
      {
       //r = r + matrix[i * colsX + j]; // original
       r = matrix[i * colsX + j - 1] - matrix[i * colsX + j + 1]; // мб надо делить на 2, но в Гонсалесе вот так(см градиент Собела/Собеля);
	   //dI = dy/dx -> у нас только вертикальные границы, поэтому считаем приращение только по x
	   //cout << "r = " << r << "\n";
        if (r > border_number)
        {
          //cout << "r = " << r << "\n";
		  //cout << "found one" << "\n";
          count_points++;
        }
		
      }
      result[i] = count_points;
	  
	  //cout << "in " << i << " row found " << result[i] << " points" << "\n";
    }
  
  /*
	cout << "result in CPU :" << "\n";
	for (int i = 0; i < rowsY; i++)
	{
		cout << result[i] << " ";
	}
	cout << '\n';
*/

    clock_t end = clock();
    elapsedTimeCPU = (double)(end-startCPU)/CLOCKS_PER_SEC;
    cout << "CPU calculating time = " << elapsedTimeCPU * 1000 << " ms\n";
    cout << "CPU memory throughput = " << elementsQuantity *sizeof(int)/elapsedTimeCPU/1024/1024/1024 << " Gb/s\n";
    
    cout << "\n";

    hipEventCreate(&startCUDA);
    hipEventCreate(&stopCUDA);

    int *dev_original_matrix, *dev_result;
    int *host_original_matrix, * host_result;

    host_original_matrix = matrix;
    host_result = new int[rowsY];
    for (int i = 0; i < rowsY; i++)
    {
      host_result[i] = 0;
    }

    CHECK( hipMalloc(&dev_original_matrix, elementsQuantity * sizeof(int)));
    CHECK( hipMalloc(&dev_result, rowsY * sizeof(int)));

    CHECK( hipMemcpy(dev_original_matrix, host_original_matrix, elementsQuantity * sizeof(int), hipMemcpyHostToDevice));
    CHECK( hipMemcpy(dev_result, host_result, rowsY * sizeof(int), hipMemcpyHostToDevice));
	
    hipEventRecord(startCUDA, 0);
    calculate<<<(rowsY + 255) / 256, 256>>>(dev_original_matrix, dev_result, elementsQuantity, rowsY, colsX, border_number);
    hipEventRecord(stopCUDA, 0);
    cout << "FINISH" << '\n';

    hipEventSynchronize(stopCUDA);
    CHECK(hipGetLastError());

    hipEventElapsedTime(&elapsedUsualTimeCUDA, startCUDA, stopCUDA);

    cout << "CUDA sum time = " << elapsedUsualTimeCUDA << " ms\n";
    cout << "CUDA memory throughput = " << elementsQuantity * sizeof(int) / elapsedUsualTimeCUDA/1024/1024/1.024 << " Gb/s\n";
    CHECK( hipMemcpy(host_result, dev_result, rowsY * sizeof(int),hipMemcpyDeviceToHost));


/*
	cout << '\n' << "host_result = " << '\n';
	printMatrix(host_result, 1, rowsY);

	cout << '\n' << "result = " << '\n';
	printMatrix(result, 1, rowsY);
*/	


    cout << "result was correct " << checkResult(host_result, result, colsX, rowsY) << "\n";
    cout << "Data size = " << (float)4 * elementsQuantity / 1024 / 1024 << "\n";

    CHECK( hipFree(dev_original_matrix));
    CHECK( hipFree(dev_result));

//}
///*
    //**********************************************************************************************
    //ХОРОШЕЕ УМНОЖЕНИЕ

    hipEventCreate(&startOptimalCUDA);
    hipEventCreate(&stopOptimalCUDA);

    int* good_host_result = new int[rowsY];
    for (int i = 0; i < rowsY; i++)
    {
      good_host_result[i] = 0; // 0
    }

    int *good_dev_result;
    CHECK( hipMalloc(&dev_original_matrix, elementsQuantity * sizeof(int)));
    CHECK( hipMalloc(&good_dev_result,rowsY * sizeof(int)));

    CHECK( hipMemcpy(dev_original_matrix, host_original_matrix, elementsQuantity * sizeof(int), hipMemcpyHostToDevice));
    CHECK( hipMemcpy(good_dev_result, good_host_result, rowsY * sizeof(int), hipMemcpyHostToDevice));
    hipEventRecord(startOptimalCUDA, 0);
    goodCalculation<<<(rowsY + 255) / 256, 256>>>(dev_original_matrix, good_dev_result, elementsQuantity, rowsY, colsX, border_number);


	//cout << '\n' << "good_host_result = " << '\n'; 
	//printMatrix(good_host_result, 1, rowsY);

	//cout << '\n' << "good_dev_result = " << '\n'; // good_dev_result пустая?
	//printMatrix(good_dev_result, 1, rowsY);


    hipEventRecord(stopOptimalCUDA, 0);
    CHECK( hipMemcpy(good_host_result, good_dev_result, rowsY * sizeof(int),hipMemcpyDeviceToHost));
    cout << ("OPTIMAL SUMMATION WAS FINISHED");
	


    hipEventElapsedTime(&elapsedOptimalTime, startOptimalCUDA, stopOptimalCUDA);

    cout << "CUDA GOOD (OPTIMAL) sum time = " << elapsedOptimalTime << " ms\n";
    cout << "CUDA GOOD (OPTIMAL) memory throughput = " << elementsQuantity * sizeof(int) / elapsedOptimalTime/1024/1024/1.024 << " Gb/s\n";




	//cout << '\n' << "good_host_result = " << '\n'; 
	//printMatrix(good_host_result, 1, rowsY);


    cout << "result was correct" <<  checkResult(good_host_result, result, colsX, rowsY) << "\n";
    cout << "Data size = " << (float)4 * elementsQuantity / 1024 / 1024 << "\n"; // float original, ok

    CHECK( hipFree(dev_original_matrix));
    CHECK( hipFree(good_dev_result));
    return 0;
}
//*/
